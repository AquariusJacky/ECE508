#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void spmvCSRKernel(float *out, int *matCols, int *matRows,
                              float *matData, float *vec, int dim) {
  // INSERT KERNEL CODE HERE

  unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < dim) {

    float result = 0.0f;
    unsigned int start = matRows[row];
    unsigned int end = matRows[row + 1];

    for (int elemIdx = start; elemIdx < end; ++elemIdx) {
      unsigned int colIdx = matCols[elemIdx];
      result += matData[elemIdx] * vec[colIdx];
    }

    out[row] = result;
  }
}

__global__ void spmvJDSKernel(float *out, int *matColStart, int *matCols,
                              int *matRowPerm, int *matRows,
                              float *matData, float *vec, int dim) {
  // INSERT KERNEL CODE HERE

  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < dim) {

    unsigned int row = matRowPerm[idx];
    float result = 0.0f;
    unsigned int rowNNZ = matRows[idx];
    for (unsigned int nzIdx = 0; nzIdx < rowNNZ; ++nzIdx) {
      unsigned int elemIdx = matColStart[nzIdx] + idx;
      unsigned int colIdx = matCols[elemIdx];
      result += matData[elemIdx] * vec[colIdx];
    }
    out[row] = result;
  }
}

static void spmvCSR(float *out, int *matCols, int *matRows, float *matData,
                    float *vec, int dim) {

  const unsigned int THREADS_PER_BLOCK = 512;
  const unsigned int numBlocks = (dim - 1) / THREADS_PER_BLOCK + 1;
  spmvCSRKernel<<<numBlocks, THREADS_PER_BLOCK>>>(out, matCols, matRows,
                                                  matData, vec, dim);
}

static void spmvJDS(float *out, int *matColStart, int *matCols,
                    int *matRowPerm, int *matRows, float *matData,
                    float *vec, int dim) {

  const unsigned int THREADS_PER_BLOCK = 512;
  const unsigned int numBlocks = (dim - 1) / THREADS_PER_BLOCK + 1;
  spmvJDSKernel<<<numBlocks, THREADS_PER_BLOCK>>>(
      out, matColStart, matCols, matRowPerm, matRows, matData, vec, dim);
}

int main(int argc, char **argv) {
  wbArg_t args;
  bool usingJDSQ;
  int *hostCSRCols;
  int *hostCSRRows;
  float *hostCSRData;
  int *hostJDSColStart;
  int *hostJDSCols;
  int *hostJDSRowPerm;
  int *hostJDSRows;
  float *hostJDSData;
  float *hostVector;
  float *hostOutput;
  int *deviceCSRCols;
  int *deviceCSRRows;
  float *deviceCSRData;
  int *deviceJDSColStart;
  int *deviceJDSCols;
  int *deviceJDSRowPerm;
  int *deviceJDSRows;
  float *deviceJDSData;
  float *deviceVector;
  float *deviceOutput;
  int dim, ncols, nrows, ndata;
  int maxRowNNZ;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  usingJDSQ = wbImport_flag(wbArg_getInputFile(args, 0)) == 1;
  hostCSRCols =
      (int *)wbImport(wbArg_getInputFile(args, 1), &ncols, "Integer");
  hostCSRRows =
      (int *)wbImport(wbArg_getInputFile(args, 2), &nrows, "Integer");
  hostCSRData =
      (float *)wbImport(wbArg_getInputFile(args, 3), &ndata, "Real");
  hostVector =
      (float *)wbImport(wbArg_getInputFile(args, 4), &dim, "Real");

  hostOutput = (float *)malloc(sizeof(float) * dim);

  wbTime_stop(Generic, "Importing data and creating memory on host");

  if (usingJDSQ) {
    CSRToJDS(dim, hostCSRRows, hostCSRCols, hostCSRData, &hostJDSRowPerm,
             &hostJDSRows, &hostJDSColStart, &hostJDSCols, &hostJDSData);
    maxRowNNZ = hostJDSRows[0];
  }

  wbTime_start(GPU, "Allocating GPU memory.");
  if (usingJDSQ) {
    hipMalloc((void **)&deviceJDSColStart, sizeof(int) * maxRowNNZ);
    hipMalloc((void **)&deviceJDSCols, sizeof(int) * ndata);
    hipMalloc((void **)&deviceJDSRowPerm, sizeof(int) * dim);
    hipMalloc((void **)&deviceJDSRows, sizeof(int) * dim);
    hipMalloc((void **)&deviceJDSData, sizeof(float) * ndata);
  } else {
    hipMalloc((void **)&deviceCSRCols, sizeof(int) * ncols);
    hipMalloc((void **)&deviceCSRRows, sizeof(int) * nrows);
    hipMalloc((void **)&deviceCSRData, sizeof(float) * ndata);
  }
  hipMalloc((void **)&deviceVector, sizeof(float) * dim);
  hipMalloc((void **)&deviceOutput, sizeof(float) * dim);
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  if (usingJDSQ) {
    hipMemcpy(deviceJDSColStart, hostJDSColStart, sizeof(int) * maxRowNNZ,
               hipMemcpyHostToDevice);
    hipMemcpy(deviceJDSCols, hostJDSCols, sizeof(int) * ndata,
               hipMemcpyHostToDevice);
    hipMemcpy(deviceJDSRowPerm, hostJDSRowPerm, sizeof(int) * dim,
               hipMemcpyHostToDevice);
    hipMemcpy(deviceJDSRows, hostJDSRows, sizeof(int) * dim,
               hipMemcpyHostToDevice);
    hipMemcpy(deviceJDSData, hostJDSData, sizeof(float) * ndata,
               hipMemcpyHostToDevice);
  } else {
    hipMemcpy(deviceCSRCols, hostCSRCols, sizeof(int) * ncols,
               hipMemcpyHostToDevice);
    hipMemcpy(deviceCSRRows, hostCSRRows, sizeof(int) * nrows,
               hipMemcpyHostToDevice);
    hipMemcpy(deviceCSRData, hostCSRData, sizeof(float) * ndata,
               hipMemcpyHostToDevice);
  }
  hipMemcpy(deviceVector, hostVector, sizeof(float) * dim,
             hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  wbTime_start(Compute, "Performing CUDA computation");
  if (usingJDSQ) {
    spmvJDS(deviceOutput, deviceJDSColStart, deviceJDSCols,
            deviceJDSRowPerm, deviceJDSRows, deviceJDSData, deviceVector,
            dim);
  } else {
    spmvCSR(deviceOutput, deviceCSRCols, deviceCSRRows, deviceCSRData,
            deviceVector, dim);
  }
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  hipMemcpy(hostOutput, deviceOutput, sizeof(float) * dim,
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceCSRCols);
  hipFree(deviceCSRRows);
  hipFree(deviceCSRData);
  hipFree(deviceVector);
  hipFree(deviceOutput);
  if (usingJDSQ) {
    hipFree(deviceJDSColStart);
    hipFree(deviceJDSCols);
    hipFree(deviceJDSRowPerm);
    hipFree(deviceJDSRows);
    hipFree(deviceJDSData);
  }
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, dim);

  free(hostCSRCols);
  free(hostCSRRows);
  free(hostCSRData);
  free(hostVector);
  free(hostOutput);
  if (usingJDSQ) {
    free(hostJDSColStart);
    free(hostJDSCols);
    free(hostJDSRowPerm);
    free(hostJDSRows);
    free(hostJDSData);
  }

  return 0;
}
